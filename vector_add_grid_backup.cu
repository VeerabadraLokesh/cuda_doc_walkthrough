
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 10000000

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    out[tid] = a[tid] + b[tid];
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Allocate device memory for a
    gpuErrchk(hipMalloc((void**)&d_a, sizeof(float) * N));
    gpuErrchk(hipMalloc((void**)&d_b, sizeof(float) * N));
    gpuErrchk(hipMalloc((void**)&d_out, sizeof(float) * N));

    // Transfer data from host to device memory
    gpuErrchk(hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice));

    int grid_blocks = N/256;

    vector_add<<<grid_blocks,256>>>(d_out, d_a, d_b, N);
    
    gpuErrchk(hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost));

    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);

    return 0;
}

